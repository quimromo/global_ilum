#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>
#include "helpers.h"
#include "random.h"
#include "device_funcs.h"

using namespace optix;

/*
struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};
*/
rtDeclareVariable(uint,			currentSample, , );
rtDeclareVariable(uint,			sqrtspp, , );
rtDeclareVariable(uint,			offset_x, , );
rtDeclareVariable(uint,			offset_y, , );
rtDeclareVariable(float,			aspect_ratio, , );
rtDeclareVariable(uint2,			screen_dim, , );
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,		 viewd, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<float3, 1>              output_buffer;
rtBuffer<float3, 1>              cumulated_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;
rtDeclareVariable(uint2, output_buffer_size, , );

//#define TIME_VIEW


RT_PROGRAM void m_pinhole_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  float2 offset = make_float2(offset_x, offset_y);
  float2 startSample = make_float2(launch_index) + offset 
	  + make_float2(
		(float)(currentSample%sqrtspp) * (1.0f/(float)sqrtspp),
		(float)(currentSample/sqrtspp) * (1.0f/(float)sqrtspp)
		);

  float2 centerSample = make_float2(1.0f/float(2 * sqrtspp));
  float2 d = ((startSample + centerSample) / make_float2(screen_dim.x, screen_dim.y)) * 2.f - 1.f;
  //float aspect_ratio = (float)launch_dim.x / launch_dim.y;
  d.x *= aspect_ratio;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W*viewd);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.contribution = make_float3(1.0f);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.seed = tea<32>((launch_index.x + offset_x) * (launch_index.y + offset_y) + (int)clock(), currentSample + (int) clock());  
  rtTrace(top_object, ray, prd);

#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else
  //output_buffer[launch_index] = make_color( prd.result );
  
  /*
  output_buffer[launch_index.y* output_buffer_size.x + launch_index.x] = make_float3 (	prd.result.z,
												prd.result.y,
												prd.result.x
											);
*/
  unsigned int buff_idx = (launch_index.y + offset_y)* screen_dim.x + launch_index.x + offset_x;
  if(currentSample == 0){
	  cumulated_buffer[buff_idx] = make_float3(0.0f);
  }
  cumulated_buffer[buff_idx] += make_float3( prd.result.z, prd.result.y, prd.result.x );
	
  float3 render_color = cumulated_buffer[buff_idx] / (currentSample + 1u);
  output_buffer[buff_idx] = make_float3 (	__saturatef( powf( render_color.z, 1.0f/2.0f) ),
											__saturatef( powf( render_color.y, 1.0f/2.0f) ),
											__saturatef( powf( render_color.x, 1.0f/2.0f) )
										);
  /*
  output_buffer[launch_index] = make_float3 (	__saturatef(prd.result.z),
												__saturatef(prd.result.y ),
												__saturatef(prd.result.x )
											);
	*/										
  //output_buffer[launch_index] = prd.result;
#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  rtPrintExceptionDetails();
  output_buffer[launch_index.y* launch_dim.x + launch_index.x] = bad_color;
}
