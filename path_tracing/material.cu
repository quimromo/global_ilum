#include "hip/hip_runtime.h"

#include "types.h"
#include "commonStructs.h"
#include "random.h"
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float3,   Ka, , );
rtDeclareVariable(float3,   Ks, , );
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   Kd, , );
rtDeclareVariable(float3,   ambient_light_color, , );
rtBuffer<BasicLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );

rtDeclareVariable(int, max_depth, , );

rtDeclareVariable(float, rnd1, , );
rtDeclareVariable(float, rnd2, , );

RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
	/*
	if(t_hit < ray.tmax && t_hit > ray.tmin){
		prd_shadow.attenuation = make_float3(0);
		rtTerminateRay();
		
	}
	*/
	prd_shadow.contribution = make_float3(0.0f);
	rtTerminateRay();
  
}

RT_PROGRAM void generic_material()
{
	// diffuse reflection
	float3 color = make_float3(0.0);
	float3 brdfLight = make_float3(0.0);
	float3 directLight = make_float3(0.0);
	float3 norm = shading_normal;
	float3 hit = ray.origin + ray.direction * t_hit;
	
	if(prd_radiance.depth < max_depth){
		unsigned int seed = prd_radiance.seed;
		float r1 = rnd(seed);
		seed = lcg(seed);
		float r2 = rnd(seed);
		float3 rd = sampleHemisphere(norm, r1, r2);
		PerRayData_radiance diffuse_refl_prd;
		diffuse_refl_prd.depth = prd_radiance.depth + 1;
		optix::Ray diffuse_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, diffuse_refl_ray, diffuse_refl_prd);
		brdfLight = diffuse_refl_prd.result;// * dot(norm, rd); 

	}
	
	for(int i = 0; i < lights.size(); ++i) {
		
		// diffuse
		BasicLight light = lights[i];
		float3 l = normalize(light.pos - hit);
		float ndl = dot(norm, l);
		if(ndl > 0.0f){
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.contribution = make_float3(1.0f);
			float ldist = length(light.pos - hit);
			optix::Ray shadow_ray( hit, l, shadow_ray_type, scene_epsilon, ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.contribution;

			if( fmaxf(light_attenuation) > 0.0f ){
				directLight = ndl*light.color;

				// specular
				float3 R = normalize(2*ndl*norm - l);
				float rdv = dot(-ray.direction, R);
				if(rdv > 0.f){
					color += Ks * pow(rdv, phong_exp) * light.color;
				}
			} 
		}
	}
	//float pathFactor = 1.0f/float(max_depth - prd_radiance.depth + 1.0f); 
	/*
	if(prd_radiance.depth < max_depth){
		PerRayData_radiance refl_prd;
		refl_prd.depth = prd_radiance.depth + 1;
		float3 refdir = ray.direction - 2*dot(norm, ray.direction)*norm;
		optix::Ray refl_ray( hit, refdir, 0, scene_epsilon );
		rtTrace(top_object, refl_ray, refl_prd);
		color += reflectivity * refl_prd.result;  
	} 	
	*/
	//prd_radiance.result = Kd * (pathFactor * directLight + (1.0f- pathFactor) * brdfLight);
	prd_radiance.result = Kd * (directLight + brdfLight);
}

RT_PROGRAM void diffuse(){
	// diffuse reflection
	float3 color = make_float3(0.0);
	float3 brdfLight = make_float3(0.0);
	float3 norm = shading_normal;
	float3 hit = ray.origin + ray.direction * t_hit;
	if(dot(norm, -ray.direction) < 0)
		norm = -norm;
	
	if(prd_radiance.depth < max_depth){
		unsigned int seed = prd_radiance.seed;
		float r1 = rnd(seed);
		float r2 = rnd(seed);
		float3 p;
		//optix::cosine_sample_hemisphere(r1, r2, p);
		p = sampleHemisphere(r1, r2);
		float3 v1, v2;
		createONB(norm, v1, v2);
		float3 rd = v1 * p.x + v2 * p.y + norm * p.z;
		
		//float3 rd = sampleHemisphere(norm, r1, r2);
		PerRayData_radiance diffuse_refl_prd;
		diffuse_refl_prd.seed = seed;
		diffuse_refl_prd.depth = prd_radiance.depth + 1;
		optix::Ray diffuse_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, diffuse_refl_ray, diffuse_refl_prd);
		brdfLight = diffuse_refl_prd.result;// * dot(norm, rd);
		color = brdfLight * Kd;
	}
	prd_radiance.result = color;
	


}

RT_PROGRAM void diffuse_and_specular(){
	// diffuse reflection
	float3 color;
	float3 reflectedLight = make_float3(0.0);
	float3 norm = shading_normal;
	float3 hit = ray.origin + ray.direction * t_hit;
	if(dot(norm, -ray.direction) < 0)
		norm = -norm;
	
	if(prd_radiance.depth < max_depth){
		float3 rd;
		unsigned int seed = prd_radiance.seed;
		float r = rnd(seed);
		float specAvg = (Ks.x + Ks.y + Ks.z) / 3;
		if(r <= specAvg){
			// specular reflection
			float3 eye = -ray.direction;
			float3 perfect_specular = 2*norm*dot(norm, eye) - eye;
			float r1 = rnd(seed);
			float r2 = rnd(seed);
			float3 p = sample_specular(phong_exp, r1, r2);
			float3 v1, v2;
			createONB(perfect_specular, v1, v2);
			rd = v1 * p.x + v2*p.y + perfect_specular*p.z;
			color = Ks;
		}
		else{
			float diffAvg = (Kd.x + Kd.y + Kd.z) / 3;
			if(r - specAvg > diffAvg){
				// absortion
				prd_radiance.result = color;
				return;

			}
			else{
				// diffuse reflection
				float r1 = rnd(seed);
				float r2 = rnd(seed);
				float3 p;
				optix::cosine_sample_hemisphere(r1, r2, p);
				//p = sampleHemisphere(r1, r2);
				float3 v1, v2;
				createONB(norm, v1, v2);
				rd = v1 * p.x + v2 * p.y + norm * p.z;
				color = Kd;
			}
		}

		PerRayData_radiance recursive_prd;
		recursive_prd.seed = seed;
		recursive_prd.depth = prd_radiance.depth + 1;
		optix::Ray recursive_ray( hit, rd, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, recursive_ray, recursive_prd);
		reflectedLight = recursive_prd.result;// * dot(norm, rd);
		prd_radiance.result = reflectedLight * color;
	}
	else prd_radiance.result = make_float3(0.0);
	


}

RT_PROGRAM void emitter(){
	
	float3 normal = geometric_normal;
	// la normal esta al rev�s!
	if(dot(-ray.direction, normal) < 0.0f) normal = -normal;
	float LnDL = dot(-ray.direction, normal);
	/*
	if( LnDL < 0){
		LnDL *= -1;
		//prd_radiance.result = Kd; 
	}
	*/
	//else prd_radiance.result = make_float3(0.0f);
	
	prd_radiance.result = Kd * LnDL;
}