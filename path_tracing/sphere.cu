#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, radius, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void sphere_isect(int)
{
	float3 ro = ray.origin - center;
	float3 rd = ray.direction;

	float a = dot(rd, rd);
	float b = 2*dot(rd, ro);
	float c = dot(ro, ro) - radius*radius;

	float discr = b*b - 4*a*c;
	if(discr < 0){
		return;
		// si no hay raiz real no hay interseccion
	} 
	
	float t0 = (-b - sqrtf(discr))/(2*a);
	if(rtPotentialIntersection(t0)){
		shading_normal = geometric_normal = (ro + rd*t0)/radius;
		if(rtReportIntersection(0))
			return;
	}

	float t1 = (-b + sqrtf(discr))/(2*a);
	if(rtPotentialIntersection(t1)){
		shading_normal = geometric_normal = (ro + rd*t1)/radius;
		rtReportIntersection(0);
	}
}

RT_PROGRAM void sphere_bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  float3 radVec = make_float3(radius);
  float3 boxmin = center - radVec;
  float3 boxmax = center + radVec;
  aabb->set(boxmin, boxmax);
}

