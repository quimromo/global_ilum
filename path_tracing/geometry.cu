#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, v0, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
//rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

RT_PROGRAM void triangle_isect(int)
{
	float3 D = ray.direction;
	float3 O = ray.origin;

	float3 e1 = v1 - v0;
	float3 e2 = v2 - v0;
	float3 T = O - v0;

	float3 P = cross(D, e2);
	float3 Q = cross(T, e1);

	float3 tuv = (1.0/dot(P, e1)) * make_float3(dot(Q, e2), dot(P, T), dot(Q, D));
	
	float t = tuv.x;
	float u = tuv.y;
	float v = tuv.z;

	if(t > 0.0001 && u >= 0.0 && v >= 0.0 && u + v <= 1.0){
		if(rtPotentialIntersection(t)){
			float3 norm = normalize(cross(e1, e2));
			if(dot(norm, -D) < 0.0){
				norm = -norm;
			}
		
			shading_normal = geometric_normal = norm;
			rtReportIntersection(0);
			return;
		}
	}
}

RT_PROGRAM void triangle_bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  float3 boxmin;
  float3 boxmax;

  boxmin.x = min(min(v0.x, v1.x), v2.x);
  boxmin.y = min(min(v0.y, v1.y), v2.y);
  boxmin.z = min(min(v0.z, v1.z), v2.z);
  
  boxmax.x = max(max(v0.x, v1.x), v2.x);
  boxmax.y = max(max(v0.y, v1.y), v2.y);
  boxmax.z = max(max(v0.z, v1.z), v2.z);
  
  aabb->set(boxmin, boxmax);
}