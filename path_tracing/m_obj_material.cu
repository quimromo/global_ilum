#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "device_funcs.h"
#include "commonStructs.h"
#include "random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


using namespace optix;

rtTextureSampler<float4, 2>   ambient_map;         // 
rtTextureSampler<float4, 2>   diffuse_map;         // Correspond to OBJ mtl params
rtTextureSampler<float4, 2>   specular_map;        //
rtTextureSampler<float4, 2> bump_map;
rtDeclareVariable(uint, usebump, , );
rtDeclareVariable(float,      phong_exp, , );          //
rtDeclareVariable(int,        illum, , );              //

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3,    tangent, attribute    tangent, );
rtDeclareVariable(float3, bi_tangent, attribute bi_tangent, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(int, is_dome, , );
rtDeclareVariable(float3, dome_emission, , );
rtDeclareVariable(unsigned int, max_direct_samples, , );
rtDeclareVariable(unsigned int, min_direct_samples, , );

rtBuffer<SphereLight, 1> spherical_lights;


RT_PROGRAM void any_hit_shadow()
{
  prd_shadow.contribution = make_float3(0.0f);
  rtTerminateRay();
}

RT_PROGRAM void diffuse()
{
  float3 direction              = ray.direction;
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 uv                     = texcoord;
  //rtPrintf("objmaterial");
  /*
  if(usebump){
	  float z = 1.0;
	  float x = make_float3(tex2D(bump_map, uv.x + 0.001, uv.y)).x - make_float3(tex2D(bump_map, uv.x - 0.001, uv.y)).x;
	  float y = make_float3(tex2D(bump_map, uv.x, uv.y + 0.001)).x - make_float3(tex2D(bump_map, uv.x, uv.y - 0.001)).x;
	  float3 bump_normal = normalize(make_float3(x,y,z));
	  //float3 obj_norm = bump_normal.x * tangent + bump_normal.y * bi_tangent + bump_normal.z * shading_normal;
	  float3 obj_norm = shading_normal + tangent * x + bi_tangent * y;
	  world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, obj_norm ) );
	  prd_radiance.result = world_shading_normal;

  }
  */
  
  
	float3 ffnormal               = normalize(faceforward( world_shading_normal, -direction, world_geometric_normal ));
 
	float3 black = make_float3(0.0f, 0.0f, 0.0f);

	float3 Kd = make_float3( tex2D( diffuse_map,  uv.x, uv.y ) );
	float3 Ks = make_float3( tex2D( specular_map, uv.x, uv.y ) );

	float prob_diff = (Kd.x + Kd.y + Kd.z) / 3.0f;
	float prob_spec = (Ks.x + Ks.y + Ks.z) / 3.0f;
	float total_prob = prob_diff + prob_spec; 
	if( total_prob > 1.0f){
		prob_diff /= total_prob;
		prob_spec /= total_prob;
		Kd /= total_prob;
		Ks /= total_prob;
		total_prob = 1.0f;
	}

  	// diffuse reflection
	float3 color = make_float3(0.0);
	float3 brdfLight = make_float3(0.0);
	float3 hit = ray.origin + ray.direction * t_hit;
	
	bool hit_light = false;
	if(prd_radiance.depth < max_depth){
		unsigned int seed = prd_radiance.seed;
		float rnd_event = rnd(seed);
		float r1 = rnd(seed);
		float r2 = rnd(seed);
		float3 p;
		float3 u, v;
		if(rnd_event <= prob_spec){
			float3 eye = -ray.direction;
			float3 perfect_specular = normalize(2.0 * ffnormal * dot(ffnormal, eye) - eye);
			createONB(perfect_specular, u, v);
			p = sample_specular2(phong_exp, r1, r2);
			float3 rd = normalize(u * p.x + v * p.y + perfect_specular * p.z);
			if(dot(rd, ffnormal) > 0.0f){
				PerRayData_radiance specular_refl_prd;
				specular_refl_prd.seed = seed;
				specular_refl_prd.depth = prd_radiance.depth + 1;
				specular_refl_prd.contribution *= Ks;
				specular_refl_prd.is_light = false;
				optix::Ray specular_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
				rtTrace(top_object, specular_refl_ray, specular_refl_prd);
				brdfLight = specular_refl_prd.result;// * dot(norm, rd);
				hit_light = specular_refl_prd.is_light;
				
				color += ( brdfLight * Ks * (phong_exp + 2) * dot(rd, ffnormal) ) / ( (phong_exp + 1) * prob_spec );
			}

		}
		else if(rnd_event <= total_prob){

			cosine_sample_hemisphere(r1, r2, p);
			createONB(ffnormal, u, v);
			float3 rd = normalize(u * p.x + v * p.y + ffnormal * p.z);
			PerRayData_radiance diffuse_refl_prd;
			diffuse_refl_prd.seed = seed;
			diffuse_refl_prd.depth = prd_radiance.depth + 1;
			diffuse_refl_prd.contribution *= Kd;
			diffuse_refl_prd.is_light = false;
			optix::Ray diffuse_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
			rtTrace(top_object, diffuse_refl_ray, diffuse_refl_prd);
			brdfLight = diffuse_refl_prd.result;// * dot(norm, rd);
			hit_light = diffuse_refl_prd.is_light;
			color += brdfLight * Kd / prob_diff;
		}

		
		
		
	}

	if(!hit_light){
		float3 perfect_specular = normalize(2.0 * ffnormal * dot(ffnormal, -ray.direction) + ray.direction);
		if(is_dome == 1){
			float3 direct_color = make_float3(0.0);
				unsigned int seed = prd_radiance.seed;
				float r1 = rnd(seed);
				float r2 = rnd(seed);
				float3 p;
				cosine_sample_hemisphere(r1, r2, p); p = normalize(p);
				//p = sampleHemisphere(r1, r2);
				float z = 1.0f - 2.0f * r1;
				float r = sqrtf( fmaxf( 0.0f, 1.0f - z*z) );
				float phi = 2.0f * r2 * PI;
				float x = r * cos(phi);
				float y = r * sin(phi);
 
				//p = normalize( make_float3(x, y, z ) );
				//float pdf = 1.0f / (4.0f * PI);
			
				/* sample solid angle
				float theta = 2.0f * PI;
				float phi = 2.0f * PI * r1;
				float alpha = acos( 1 - (1- cosf(theta) ) * r2 );
				float x = sin(alpha)*cosf(phi);
				float y = sin(alpha)*sinf(phi);
				float z = -cos(alpha);
				p = make_float3(x, y, z);
				*/

				float3 v1, v2;
				createONB(ffnormal, v1, v2);
				float3 rd = normalize(v1 * p.x + v2 * p.y + ffnormal * p.z);
				//if(dot(rd, ffnormal) < 0.0f) rd *= -1;
				PerRayData_shadow shadow_prd;
				shadow_prd.contribution = make_float3(1.0f);
				Ray shadow_ray = Ray( hit, rd, shadow_ray_type, scene_epsilon);
				rtTrace(top_object, shadow_ray, shadow_prd);
				direct_color +=  shadow_prd.contribution;// * dot(rd, ffnormal);

			
			direct_color *= Kd * dome_emission;
			color += direct_color;
		}
		
		for(int i = 0; i < spherical_lights.size(); ++i){
			float3 light_dir = spherical_lights[i].center - hit;
			float dist2 = dot(light_dir, light_dir);
			float radius2 = spherical_lights[i].radius * spherical_lights[i].radius;
			if(dist2 - radius2 < scene_epsilon){
				continue;
			}
			unsigned int seed = prd_radiance.seed;
			float cos_theta_max = sqrtf(1 - radius2/dist2);
			float inv_pdf = 2.0f * PI * (1.0f - cos_theta_max);
			
			float r1 = rnd(seed);
			float r2 = rnd(seed);

			float cos_theta = 1 + r1 * (cos_theta_max - 1);
			float sin2theta = 1 - cos_theta * cos_theta;
			float sin_theta = sqrtf(sin2theta);
			float sin_phi = sinf(2 * PI * r2);
			float cos_phi = cosf(2 * PI * r2);
			//float3 dir = make_float3( sqrtf(sin2theta), cos_theta, 2 * PI * r2);
			float3 w = normalize(light_dir);
			float3 u, v;
			createONB(w, u, v);
			float3 dir = make_float3(u.x * cos_phi * sin_theta + v.x * sin_phi * sin_theta + w.x * cos_theta,
									 u.y * cos_phi * sin_theta + v.y * sin_phi * sin_theta + w.y * cos_theta,
									 u.z * cos_phi * sin_theta + v.z * sin_phi * sin_theta + w.z * cos_theta
									);

			if(dot(dir, ffnormal) < 0) continue;
			PerRayData_shadow shadow_prd;
			shadow_prd.contribution = spherical_lights[i].emission;
			float delta = sqrtf(radius2 - sin2theta * dist2);
			Ray shadow_ray = Ray( hit, dir, shadow_ray_type, scene_epsilon, cos_theta * length(light_dir) - delta );
			rtTrace(top_object, shadow_ray, shadow_prd);
			color += inv_pdf * shadow_prd.contribution * dot(dir, ffnormal) * (Kd + Ks * fmaxf(dot(dir, perfect_specular), 0.0f));

		}
		

	}

	prd_radiance.result = color;
	//prd_radiance.dist = fabs(t_hit) / 10.0f;
  
	
  //prd_radiance.result = Kd;
}

rtDeclareVariable(float3, emission, , );

RT_PROGRAM void emitter(){
	
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal               = normalize(faceforward( world_shading_normal, -ray.direction, world_geometric_normal ));
	// la normal esta al rev�s!
	float LnDL = dot(-ray.direction, ffnormal);
	//if( LnDL < 0){

		prd_radiance.result = emission;// * LnDL;
		//prd_radiance.dist = t_hit/10.0f;
		prd_radiance.is_light = true;
	//}
	//else prd_radiance.result = make_float3(0.0f);
	
	//prd_radiance.result = Kd;
}
