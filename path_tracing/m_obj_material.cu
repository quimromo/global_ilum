#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "types.h"
#include "commonStructs.h"
#include "random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


using namespace optix;

rtTextureSampler<float4, 2>   ambient_map;         // 
rtTextureSampler<float4, 2>   diffuse_map;         // Correspond to OBJ mtl params
rtTextureSampler<float4, 2>   specular_map;        //
rtTextureSampler<float4, 2> bump_map;
rtDeclareVariable(uint, usebump, , );
rtDeclareVariable(float,      phong_exp, , );          //
rtDeclareVariable(int,        illum, , );              //

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3,    tangent, attribute    tangent, );
rtDeclareVariable(float3, bi_tangent, attribute bi_tangent, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(int, max_depth, , );


RT_PROGRAM void any_hit_shadow()
{
  //phongShadowed();
}

RT_PROGRAM void diffuse()
{
  float3 direction              = ray.direction;
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 uv                     = texcoord;
  //rtPrintf("objmaterial");
  
  if(usebump){
	  float z = 1.0;
	  float x = make_float3(tex2D(bump_map, uv.x + 0.001, uv.y)).x - make_float3(tex2D(bump_map, uv.x - 0.001, uv.y)).x;
	  float y = make_float3(tex2D(bump_map, uv.x, uv.y + 0.001)).x - make_float3(tex2D(bump_map, uv.x, uv.y - 0.001)).x;
	  float3 bump_normal = normalize(make_float3(x,y,z));
	  //float3 obj_norm = bump_normal.x * tangent + bump_normal.y * bi_tangent + bump_normal.z * shading_normal;
	  float3 obj_norm = shading_normal + tangent * x + bi_tangent * y;
	  world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, obj_norm ) );
	  prd_radiance.result = world_shading_normal;

  }
  
  else{
  
  float3 ffnormal               = normalize(faceforward( world_shading_normal, -direction, world_geometric_normal ));
  

  float3 black = make_float3(0.0f, 0.0f, 0.0f);

  // grab values from textures
  // support only MTL illumination modes 0-3 (Ks is for now used as reflectivity)
  // ( see http://local.wasp.uwa.edu.au/~pbourke/dataformats/mtl/ )
  float3 Kd =                       make_float3( tex2D( diffuse_map,  uv.x, uv.y ) );
  //float3 Ka = (illum < 1) ? black : make_float3( tex2D( ambient_map,  uv.x, uv.y ) );
  //float3 Ks = (illum < 2) ? black : make_float3( tex2D( specular_map, uv.x, uv.y ) );
  //float3 Kr = (illum < 3) ? black : Ks;
  
  	// diffuse reflection
	float3 color = make_float3(0.0);
	float3 brdfLight = make_float3(0.0);
	float3 hit = ray.origin + ray.direction * t_hit;
	
	if(prd_radiance.depth < max_depth){
		unsigned int seed = prd_radiance.seed;
		float r1 = rnd(seed);
		float r2 = rnd(seed);
		float3 p;
		cosine_sample_hemisphere(r1, r2, p);
		//p = sampleHemisphere(r1, r2);
		
		float3 v1, v2;
		createONB(ffnormal, v1, v2);
		float3 rd = normalize(v1 * p.x + v2 * p.y + ffnormal * p.z);
		
		//float3 rd = sampleHemisphere(norm, r1, r2);
		PerRayData_radiance diffuse_refl_prd;
		diffuse_refl_prd.seed = seed;
		diffuse_refl_prd.depth = prd_radiance.depth + 1;
		optix::Ray diffuse_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, diffuse_refl_ray, diffuse_refl_prd);
		brdfLight = diffuse_refl_prd.result;// * dot(norm, rd);
		color = brdfLight * Kd * dot(ffnormal, rd);// / (diffuse_refl_prd.dist*diffuse_refl_prd.dist);
	}
	prd_radiance.result = color;
	prd_radiance.dist = t_hit;
  }
	
  //prd_radiance.result = Kd;
}

rtDeclareVariable(float3, emission, , );

RT_PROGRAM void emitter(){
	
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal               = normalize(faceforward( world_shading_normal, -ray.direction, world_geometric_normal ));
	// la normal esta al rev�s!
	float LnDL = dot(-ray.direction, ffnormal);
	//if( LnDL < 0){

		prd_radiance.result = emission * LnDL;
		prd_radiance.dist = t_hit;
	//}
	//else prd_radiance.result = make_float3(0.0f);
	
	//prd_radiance.result = Kd;
}
