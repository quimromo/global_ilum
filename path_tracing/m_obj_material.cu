#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "types.h"
#include "commonStructs.h"
#include "random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


using namespace optix;

rtTextureSampler<float4, 2>   ambient_map;         // 
rtTextureSampler<float4, 2>   diffuse_map;         // Correspond to OBJ mtl params
rtTextureSampler<float4, 2>   specular_map;        //
rtTextureSampler<float4, 2> bump_map;
rtDeclareVariable(uint, usebump, , );
rtDeclareVariable(float,      phong_exp, , );          //
rtDeclareVariable(int,        illum, , );              //

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3,    tangent, attribute    tangent, );
rtDeclareVariable(float3, bi_tangent, attribute bi_tangent, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(int, is_dome, , );
rtDeclareVariable(float3, dome_emission, , );
rtDeclareVariable(unsigned int, max_direct_samples, , );
rtDeclareVariable(unsigned int, min_direct_samples, , );

rtBuffer<SphereLight, 1> spherical_lights;


RT_PROGRAM void any_hit_shadow()
{
  prd_shadow.contribution = make_float3(0.0f);
  rtTerminateRay();
}

RT_PROGRAM void diffuse()
{
  float3 direction              = ray.direction;
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 uv                     = texcoord;
  //rtPrintf("objmaterial");
  /*
  if(usebump){
	  float z = 1.0;
	  float x = make_float3(tex2D(bump_map, uv.x + 0.001, uv.y)).x - make_float3(tex2D(bump_map, uv.x - 0.001, uv.y)).x;
	  float y = make_float3(tex2D(bump_map, uv.x, uv.y + 0.001)).x - make_float3(tex2D(bump_map, uv.x, uv.y - 0.001)).x;
	  float3 bump_normal = normalize(make_float3(x,y,z));
	  //float3 obj_norm = bump_normal.x * tangent + bump_normal.y * bi_tangent + bump_normal.z * shading_normal;
	  float3 obj_norm = shading_normal + tangent * x + bi_tangent * y;
	  world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, obj_norm ) );
	  prd_radiance.result = world_shading_normal;

  }
  */
  
  
  float3 ffnormal               = normalize(faceforward( world_shading_normal, -direction, world_geometric_normal ));
  

  float3 black = make_float3(0.0f, 0.0f, 0.0f);

  // grab values from textures
  // support only MTL illumination modes 0-3 (Ks is for now used as reflectivity)
  // ( see http://local.wasp.uwa.edu.au/~pbourke/dataformats/mtl/ )
  float3 Kd =                       make_float3( tex2D( diffuse_map,  uv.x, uv.y ) );
  //float3 Ka = (illum < 1) ? black : make_float3( tex2D( ambient_map,  uv.x, uv.y ) );
  //float3 Ks = (illum < 2) ? black : make_float3( tex2D( specular_map, uv.x, uv.y ) );
  //float3 Kr = (illum < 3) ? black : Ks;
  
  	// diffuse reflection
	float3 color = make_float3(0.0);
	float3 brdfLight = make_float3(0.0);
	float3 hit = ray.origin + ray.direction * t_hit;
	
	bool hit_light = false;
	if(prd_radiance.depth < max_depth){
		unsigned int seed = prd_radiance.seed;
		float r1 = rnd(seed);
		float r2 = rnd(seed);
		float3 p;
		cosine_sample_hemisphere(r1, r2, p);
		//p = sampleHemisphere(r1, r2);
		
		float3 v1, v2;
		createONB(ffnormal, v1, v2);
		float3 rd = normalize(v1 * p.x + v2 * p.y + ffnormal * p.z);
		
		//float3 rd = sampleHemisphere(norm, r1, r2);
		PerRayData_radiance diffuse_refl_prd;
		diffuse_refl_prd.seed = seed;
		diffuse_refl_prd.depth = prd_radiance.depth + 1;
		diffuse_refl_prd.contribution *= Kd;
		diffuse_refl_prd.is_light = false;
		optix::Ray diffuse_refl_ray( hit, rd, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, diffuse_refl_ray, diffuse_refl_prd);
		brdfLight = diffuse_refl_prd.result;// * dot(norm, rd);
		color += brdfLight * Kd;// / diffuse_refl_prd.dist;// * diffuse_refl_prd.dist); // * dot(ffnormal, rd);// / (diffuse_refl_prd.dist*diffuse_refl_prd.dist);
		hit_light = diffuse_refl_prd.is_light;
	}

	if(!hit_light){

		if(is_dome){
			float3 direct_color = make_float3(0.0);
			float contrib = (prd_radiance.contribution.x + prd_radiance.contribution.y + prd_radiance.contribution.z) / 3.0f;
			unsigned int num_samples = (unsigned int)fmaxf(contrib * (float)max_direct_samples, (float)min_direct_samples);
			//unsigned int num_samples = 20u;
			for(unsigned int i = 0; i < num_samples; ++i){
				unsigned int seed = prd_radiance.seed;
				float r1 = rnd(seed);
				float r2 = rnd(seed);
				float3 p;
				//cosine_sample_hemisphere(r1, r2, p);
				//p = sampleHemisphere(r1, r2);
				float z = 1.0f - 2.0f * r1;
				float r = sqrtf( fmaxf( 0.0f, 1.0f - z*z) );
				float phi = 2.0f * r2 * PI;
				float x = r * cos(phi);
				float y = r * sin(phi);
 
				p = normalize( make_float3(x, y, z ) );
				float pdf = 1.0f / (4.0f * PI);
			
				/* sample solid angle
				float theta = 2.0f * PI;
				float phi = 2.0f * PI * r1;
				float alpha = acos( 1 - (1- cosf(theta) ) * r2 );
				float x = sin(alpha)*cosf(phi);
				float y = sin(alpha)*sinf(phi);
				float z = -cos(alpha);
				p = make_float3(x, y, z);
				*/

				float3 v1, v2;
				createONB(ffnormal, v1, v2);
				float3 rd = normalize(v1 * p.x + v2 * p.y + ffnormal * p.z);
				if(dot(rd, ffnormal) < 0.0f) rd *= -1;
				PerRayData_shadow shadow_prd;
				shadow_prd.contribution = make_float3(1.0f);
				Ray shadow_ray = Ray( hit, rd, shadow_ray_type, scene_epsilon);
				rtTrace(top_object, shadow_ray, shadow_prd);
				direct_color +=  shadow_prd.contribution * dot(rd, ffnormal);

			}
			direct_color *= Kd * 2.0f * PI * dome_emission;
			direct_color /= num_samples;
			color += direct_color;
		}

		for(int i = 0; i < spherical_lights.size(); ++i){
			float3 light_dir = spherical_lights[i] - hit;
			float dist2 = dot(light_dir, light_dir);
			float radius2 = spherical_lights[i].radius * spherical_lights[i].radius;
			if(dist2 - radius2 < scene_epsilon){
				continue;
			}
			float cos_theta_max = sqrtf(1 - radius2/dist2);
			float pdf = 1.0f / (2.0f * PI * (1.0f - cos_theta_max));



		}


	}

	prd_radiance.result = color;
	//prd_radiance.dist = fabs(t_hit) / 10.0f;
  
	
  //prd_radiance.result = Kd;
}

rtDeclareVariable(float3, emission, , );

RT_PROGRAM void emitter(){
	
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal               = normalize(faceforward( world_shading_normal, -ray.direction, world_geometric_normal ));
	// la normal esta al rev�s!
	float LnDL = dot(-ray.direction, ffnormal);
	//if( LnDL < 0){

		prd_radiance.result = emission;// * LnDL;
		//prd_radiance.dist = t_hit/10.0f;
		prd_radiance.is_light = true;
	//}
	//else prd_radiance.result = make_float3(0.0f);
	
	//prd_radiance.result = Kd;
}
